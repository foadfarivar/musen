#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2020, MUSEN Development Team. All rights reserved.
   This file is part of MUSEN framework http://msolids.net/musen.
   See LICENSE file for license and warranty information. */

#include "ModelPWSimpleViscoelastic.cuh"
#include "ModelPWSimpleViscoElastic.h"
#include <>

__constant__ SPBC PBC;
__constant__ double m_vConstantModelParameters[2];

void CModelPWSimpleViscoElastic::SetParametersGPU(const std::vector<double>& _parameters, const SPBC& _pbc)
{
	CUDA_MEMCOPY_TO_SYMBOL(m_vConstantModelParameters, *_parameters.data(), sizeof(double) * _parameters.size());
	CUDA_MEMCOPY_TO_SYMBOL(PBC, _pbc, sizeof(SPBC));
}

void CModelPWSimpleViscoElastic::CalculatePWForceGPU(double _time, double _timeStep, const SInteractProps _interactProps[], const SGPUParticles& _particles, const SGPUWalls& _walls, SGPUCollisions& _collisions)
{
	CUDA_KERNEL_ARGS2_DEFAULT(CUDA_CalcPWForce_VE_kernel,
		_particles.Coords,
		_particles.Radii,
		_particles.Vels,
		_particles.Forces,
		_particles.Moments,

		_walls.Vels,
		_walls.RotCenters,
		_walls.RotVels,
		_walls.NormalVectors,
		_walls.Forces,

		_collisions.ActiveCollisionsNum,
		_collisions.ActivityIndices,
		_collisions.ContactVectors,  // interpreted as Contact Point
		_collisions.SrcIDs,
		_collisions.DstIDs,
		_collisions.VirtualShifts,

		_collisions.TotalForces
	);
}

void __global__ CUDA_CalcPWForce_VE_kernel(
	const CVector3	_partCoords[],
	const double	_partRadii[],
	const CVector3	_partVels[],
	CVector3		_partForces[],
	CVector3		_partMoments[],

	const CVector3	_wallVels[],
	const CVector3	_wallRotCenters[],
	const CVector3	_wallRotVels[],
	const CVector3	_wallNormalVecs[],
	CVector3        _wallForces[],

	const unsigned*	_collActiveCollisionsNum,
	const unsigned	_collActivityIndices[],
	const CVector3	_collContactPoints[],
	const unsigned	_collSrcIDs[],
	const unsigned	_collDstIDs[],
	const uint8_t   _collVirtShifts[],

	CVector3 _collTotalForces[]
)
{
	for (unsigned iActivColl = blockIdx.x * blockDim.x + threadIdx.x; iActivColl < *_collActiveCollisionsNum; iActivColl += blockDim.x * gridDim.x)
	{
		const unsigned iColl      = _collActivityIndices[iActivColl];
		const unsigned iWall      = _collSrcIDs[iColl];
		const unsigned iPart      = _collDstIDs[iColl];
		const CVector3 normVector = _wallNormalVecs[iWall];

		// model parameters
		const double Kn = m_vConstantModelParameters[0];
		const double mu = m_vConstantModelParameters[1];

		const CVector3 rc     = GPU_GET_VIRTUAL_COORDINATE(_partCoords[iPart]) - _collContactPoints[iColl];
		const double   rcLen  = rc.Length();
		const CVector3 rcNorm = rc / rcLen;

		// normal overlap
		const double normOverlap = _partRadii[iPart] - rcLen;
		if (normOverlap < 0) continue;

		// normal and tangential relative velocity
		const CVector3 rotVel   = !_wallRotVels[iWall].IsZero() ? (_collContactPoints[iColl] - _wallRotCenters[iWall]) * _wallRotVels[iWall] : CVector3{ 0 };
		const CVector3 relVel   = _partVels[iPart] - _wallVels[iWall] + rotVel;
		const double normRelVel = DotProduct(normVector, relVel);

		// normal force with damping
		const double normContactForceLen = normOverlap * Kn * fabs(DotProduct(rcNorm, normVector));
		const double normDampingForceLen = -mu * normRelVel;
		const CVector3 normForce = normVector * (normContactForceLen + normDampingForceLen);

		// store results in collision
		_collTotalForces[iColl] = normForce;

		// apply forces and moments
		CUDA_VECTOR3_ATOMIC_ADD(_partForces[iPart], normForce);
		CUDA_VECTOR3_ATOMIC_SUB(_wallForces[iWall], normForce);
	}
}