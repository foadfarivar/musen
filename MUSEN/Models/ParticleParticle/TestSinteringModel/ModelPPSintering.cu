#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2020, MUSEN Development Team. All rights reserved.
   This file is part of MUSEN framework http://msolids.net/musen.
   See LICENSE file for license and warranty information. */

#include "ModelPPSintering.cuh"
#include "ModelPPSintering.h"
#include <>

__constant__ double m_vConstantModelParameters[2];

void CModelPPSintering::SetParametersGPU(const std::vector<double>& _parameters, const SPBC& _pbc)
{
	CUDA_MEMCOPY_TO_SYMBOL(m_vConstantModelParameters, *_parameters.data(), sizeof(double) * _parameters.size());
}

void CModelPPSintering::CalculatePPForceGPU(double _time, double _timeStep, const SInteractProps _interactProps[], const SGPUParticles& _particles, SGPUCollisions& _collisions)
{
	CUDA_KERNEL_ARGS2_DEFAULT(CUDA_CalcPPForce_S_kernel,
		_interactProps,

		_particles.Vels,
		_particles.Forces,

		_collisions.ActiveCollisionsNum,
		_collisions.ActivityIndices,
		_collisions.InteractPropIDs,
		_collisions.SrcIDs,
		_collisions.DstIDs,
		_collisions.EquivRadii,
		_collisions.NormalOverlaps,
		_collisions.ContactVectors
	);
}

void __global__ CUDA_CalcPPForce_S_kernel(
	const SInteractProps	_interactProps[],
	const CVector3	_partVels[],
	CVector3	_partForces[],

	const unsigned*	_collActiveCollisionsNum,
	const unsigned	_collActivityIndices[],
	const uint16_t	_collInteractPropIDs[],
	const unsigned	_collSrcIDs[],
	const unsigned	_collDstIDs[],
	const double	_collEquivRadii[],
	const double	_collNormalOverlaps[],
	const CVector3  _collContactVectors[]
)
{
	for (unsigned iActivColl = blockIdx.x * blockDim.x + threadIdx.x; iActivColl < *_collActiveCollisionsNum; iActivColl += blockDim.x * gridDim.x)
	{
		const unsigned iColl = _collActivityIndices[iActivColl];
		const unsigned iSrcPart = _collSrcIDs[iColl];
		const unsigned iDstPart = _collDstIDs[iColl];
		const double dEquivRadius = _collEquivRadii[iColl];

		CVector3 vNormalVector = _collContactVectors[iColl].Normalized();

		//obtain velocities
		CVector3 vRelVel           = _partVels[iSrcPart] - _partVels[iDstPart];
		CVector3 vRelVelNormal     = vNormalVector * DotProduct(vNormalVector, vRelVel);
		CVector3 vRelVelTangential = vRelVel - vRelVelNormal;

		//Bouvard and McMeeking's model
		const double dSquaredContactRadius = 4 * dEquivRadius * _collNormalOverlaps[iColl];

		// calculate forces
		const CVector3 vSinteringForce = vNormalVector * 1.125 * PI * 2 * dEquivRadius * _interactProps[_collInteractPropIDs[iColl]].dEquivSurfaceEnergy;
		const CVector3 vViscousForce = vRelVelNormal * (-PI * pow(dSquaredContactRadius, 2.0) / 8 / m_vConstantModelParameters[0]);
		const CVector3 vTangentialForce = vRelVelTangential * (-m_vConstantModelParameters[1] * PI * dSquaredContactRadius * pow(2 * dEquivRadius, 2.0) / 8 / m_vConstantModelParameters[0]);
		const CVector3 vTotalForce = vSinteringForce + vViscousForce + vTangentialForce;

		// apply forces
		CUDA_VECTOR3_ATOMIC_ADD(_partForces[iSrcPart], vTotalForce);
		CUDA_VECTOR3_ATOMIC_SUB(_partForces[iDstPart], vTotalForce);
	}
}