#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2020, MUSEN Development Team. All rights reserved.
   This file is part of MUSEN framework http://msolids.net/musen.
   See LICENSE file for license and warranty information. */

#include "ModelPPSimpleViscoElastic.cuh"
#include "ModelPPSimpleViscoElastic.h"
#include <>
#include <thrust/device_vector.h>

__constant__ double m_vConstantModelParameters[2];

void CModelPPSimpleViscoElastic::SetParametersGPU(const std::vector<double>& _parameters, const SPBC& _pbc)
{
	CUDA_MEMCOPY_TO_SYMBOL(m_vConstantModelParameters, *_parameters.data(), sizeof(double) * _parameters.size());
}

void CModelPPSimpleViscoElastic::CalculatePPForceGPU(double _time, double _timeStep, const SInteractProps _interactProps[], const SGPUParticles& _particles, SGPUCollisions& _collisions)
{
	CUDA_KERNEL_ARGS2_DEFAULT(CUDA_CalcPPForce_VE_kernel,
		_particles.Vels,
		_particles.Forces,

		_collisions.ActiveCollisionsNum,
		_collisions.ActivityIndices,
		_collisions.SrcIDs,
		_collisions.DstIDs,
		_collisions.NormalOverlaps,
		_collisions.ContactVectors,

		_collisions.TotalForces
	);
}

void __global__ CUDA_CalcPPForce_VE_kernel(
	const CVector3	_partVels[],
	CVector3		_partForces[],

	const unsigned*	_collActiveCollisionsNum,
	const unsigned	_collActivityIndices[],
	const unsigned	_collSrcIDs[],
	const unsigned	_collDstIDs[],
	const double	_collNormalOverlaps[],
	const CVector3	_collContactVectors[],

	CVector3 _collTotalForces[]
)
{
	for (unsigned iActivColl = blockIdx.x * blockDim.x + threadIdx.x; iActivColl < *_collActiveCollisionsNum; iActivColl += blockDim.x * gridDim.x)
	{
		const unsigned iColl    = _collActivityIndices[iActivColl];
		const unsigned iSrcPart = _collSrcIDs[iColl];
		const unsigned iDstPart = _collDstIDs[iColl];

		const double dKn = m_vConstantModelParameters[0];
		const double dMu = m_vConstantModelParameters[1];

		const double dNormalOverlap  = _collNormalOverlaps[iColl];
		const CVector3 vNormalVector = _collContactVectors[iColl].Normalized();

		// relative velocity (normal and tangential)
		const CVector3 vRelVelocity = _partVels[iDstPart] - _partVels[iSrcPart];
		const double dRelVelNormal  = DotProduct(vNormalVector, vRelVelocity);

		// normal force with damping
		const double dDampingForce = -dMu * dRelVelNormal;

		// calculate forces
		const double dNormalForce = -dNormalOverlap * dKn;
		const CVector3 vTotalForce = (dNormalForce + dDampingForce) * vNormalVector;

		// store results in collision
		_collTotalForces[iColl] = vTotalForce;

		// apply forces
		CUDA_VECTOR3_ATOMIC_ADD(_partForces[iSrcPart], vTotalForce);
		CUDA_VECTOR3_ATOMIC_SUB(_partForces[iDstPart], vTotalForce);
	}
}