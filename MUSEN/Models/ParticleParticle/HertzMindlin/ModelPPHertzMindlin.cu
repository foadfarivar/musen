#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2020, MUSEN Development Team. All rights reserved.
   This file is part of MUSEN framework http://msolids.net/musen.
   See LICENSE file for license and warranty information. */

#include "ModelPPHertzMindlin.cuh"
#include "ModelPPHertzMindlin.h"
#include <>

void CModelPPHertzMindlin::SetParametersGPU(const std::vector<double>& _parameters, const SPBC& _pbc)
{
}

void CModelPPHertzMindlin::CalculatePPForceGPU(double _time, double _timeStep, const SInteractProps _interactProps[], const SGPUParticles& _particles, SGPUCollisions& _collisions)
{
	CUDA_KERNEL_ARGS2_DEFAULT(CUDA_CalcPPForce_HM_kernel,
		_timeStep,
		_interactProps,

		_particles.AnglVels,
		_particles.Radii,
		_particles.Vels,
		_particles.Forces,
		_particles.Moments,

		_collisions.ActiveCollisionsNum,
		_collisions.ActivityIndices,
		_collisions.InteractPropIDs,
		_collisions.SrcIDs,
		_collisions.DstIDs,
		_collisions.EquivMasses,
		_collisions.EquivRadii,
		_collisions.NormalOverlaps,
		_collisions.ContactVectors,

		_collisions.TangOverlaps
	);
}

void __global__ CUDA_CalcPPForce_HM_kernel(
	double					_timeStep,
	const SInteractProps	_interactProps[],

	const CVector3	_partAnglVels[],
	const double	_partRadii[],
	const CVector3	_partVels[],
	CVector3		_partForces[],
	CVector3		_partMoments[],

	const unsigned*	_collActiveCollisionsNum,
	const unsigned	_collActivityIndices[],
	const uint16_t	_collInteractPropIDs[],
	const unsigned	_collSrcIDs[],
	const unsigned	_collDstIDs[],
	const double	_collEquivMasses[],
	const double	_collEquivRadii[],
	const double	_collNormalOverlaps[],
	const CVector3	_collContactVectors[],

	CVector3 _collTangOverlaps[]
)
{
	for (unsigned iActivColl = blockIdx.x * blockDim.x + threadIdx.x; iActivColl < *_collActiveCollisionsNum; iActivColl += blockDim.x * gridDim.x)
	{
		unsigned iColl            = _collActivityIndices[iActivColl];
		unsigned iSrcPart         = _collSrcIDs[iColl];
		unsigned iDstPart         = _collDstIDs[iColl];
		SInteractProps prop       = _interactProps[_collInteractPropIDs[iColl]];
		double dNormalOverlap     = _collNormalOverlaps[iColl];
		double dEquivMass         = _collEquivMasses[iColl];
		const CVector3 srcAnglVel = _partAnglVels[iSrcPart];
		const CVector3 dstAnglVel = _partAnglVels[iDstPart];
		double dPartSrcRadius     = _partRadii[iSrcPart];
		double dPartDstRadius     = _partRadii[iDstPart];

		const CVector3 vContactVector = _collContactVectors[iColl];
		const CVector3 vRcSrc         = vContactVector * ( dPartSrcRadius / (dPartSrcRadius + dPartDstRadius));
		const CVector3 vRcDst         = vContactVector * (-dPartDstRadius / (dPartSrcRadius + dPartDstRadius));
		const CVector3 vNormalVector  = vContactVector.Normalized();

		// relative velocity (normal and tangential)
		const CVector3 vRelVel       = _partVels[iDstPart] + dstAnglVel * vRcDst - (_partVels[iSrcPart] + srcAnglVel * vRcSrc);
		const double   dRelVelNormal = DotProduct(vNormalVector, vRelVel);
		const CVector3 vRelVelNormal = dRelVelNormal * vNormalVector;
		const CVector3 vRelVelTang   = vRelVel - vRelVelNormal;

		// normal and tangential overlaps
		CVector3 vDeltaTangOverlap = vRelVelTang * _timeStep;

		// a set of parameters for fast access
		double dTemp2 = sqrt(_collEquivRadii[iColl] * dNormalOverlap);

		// normal force with damping
		double Kn = 2 * prop.dEquivYoungModulus * dTemp2;
		const double dDampingForce = -1.8257 * prop.dAlpha * dRelVelNormal * sqrt(Kn * dEquivMass);
		const double dNormalForce = -dNormalOverlap * Kn * 2. / 3.;

		// increment of tangential force with damping
		double Kt = 8 * prop.dEquivShearModulus * dTemp2;
		CVector3 vDampingTangForce = vRelVelTang * (-1.8257 * prop.dAlpha * sqrt(Kt * dEquivMass));

		// rotate old tangential force
		CVector3 vOldTangOverlap = _collTangOverlaps[iColl];
		CVector3 vTangOverlap = vOldTangOverlap - vNormalVector * DotProduct(vNormalVector, vOldTangOverlap);
		double dTangOverlapSqrLen = vTangOverlap.SquaredLength();
		if (dTangOverlapSqrLen > 0)
			vTangOverlap = vTangOverlap * vOldTangOverlap.Length() / sqrt(dTangOverlapSqrLen);
		vTangOverlap += vDeltaTangOverlap;

		CVector3 vTangForce = vTangOverlap * Kt;

		// check slipping condition
		double dNewTangForce = vTangForce.Length();
		if (dNewTangForce > prop.dSlidingFriction * fabs(dNormalForce))
		{
			vTangForce *= prop.dSlidingFriction * fabs(dNormalForce) / dNewTangForce;
			vTangOverlap = vTangForce / Kt;
		}
		else
			vTangForce += vDampingTangForce;

		// calculate rolling torque
		const CVector3 vRollingTorque1 = srcAnglVel.IsSignificant() ? // if it is not zero, but small enough, its Length() can turn into zero and division fails
			srcAnglVel * (-1 * prop.dRollingFriction * fabs(dNormalForce) * dPartSrcRadius / srcAnglVel.Length()) : CVector3{ 0 };
		const CVector3 vRollingTorque2 = dstAnglVel.IsSignificant() ? // if it is not zero, but small enough, its Length() can turn into zero and division fails
			dstAnglVel * (-1 * prop.dRollingFriction * fabs(dNormalForce) * dPartDstRadius / dstAnglVel.Length()) : CVector3{ 0 };

		// store results in collision
		_collTangOverlaps[iColl] = vTangOverlap;

		// calculate moments and forces
		const CVector3 vTotalForce    = vNormalVector * (dNormalForce + dDampingForce) + vTangForce;
		const CVector3 vResultMoment1 = vNormalVector * vTangForce * dPartSrcRadius + vRollingTorque1;
		const CVector3 vResultMoment2 = vNormalVector * vTangForce * dPartDstRadius + vRollingTorque2;

		// apply moments and forces
		CUDA_VECTOR3_ATOMIC_ADD(_partForces[iSrcPart], vTotalForce);
		CUDA_VECTOR3_ATOMIC_SUB(_partForces[iDstPart], vTotalForce);
		CUDA_VECTOR3_ATOMIC_ADD(_partMoments[iSrcPart], vResultMoment1);
		CUDA_VECTOR3_ATOMIC_ADD(_partMoments[iDstPart], vResultMoment2);
	}
}