#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2020, MUSEN Development Team. All rights reserved.
   This file is part of MUSEN framework http://msolids.net/musen.
   See LICENSE file for license and warranty information. */

#include "ModelPPHertzMindlinVdW.cuh"
#include "ModelPPHertzMindlinVdW.h"
#include <>

void CModelPPHertzMindlinVdW::SetParametersGPU(const std::vector<double>& _parameters, const SPBC& _pbc)
{
}

void CModelPPHertzMindlinVdW::CalculatePPForceGPU(double _time, double _timeStep, const SInteractProps _interactProps[], const SGPUParticles& _particles, SGPUCollisions& _collisions)
{
	CUDA_KERNEL_ARGS2_DEFAULT(CUDA_CalcPPForce_HMVDW_kernel,
		_timeStep,
		_interactProps,

		_particles.AnglVels,
		_particles.Radii,
		_particles.Vels,
		_particles.Coords,
		_particles.Forces,
		_particles.Moments,

		_collisions.ActiveCollisionsNum,
		_collisions.ActivityIndices,
		_collisions.InteractPropIDs,
		_collisions.SrcIDs,
		_collisions.DstIDs,
		_collisions.EquivMasses,
		_collisions.EquivRadii,
		_collisions.ContactVectors,

		_collisions.TangOverlaps,
		_collisions.TotalForces
	);
}

void __global__ CUDA_CalcPPForce_HMVDW_kernel(
	double				 _timeStep,
	const SInteractProps _interactProps[],

	const CVector3	_partAnglVels[],
	const double	_partRadii[],
	const CVector3	_partVels[],
	const CVector3	_partCoords[],
	CVector3		_partForces[],
	CVector3		_partMoments[],

	const unsigned*	_collActiveCollisionsNum,
	const unsigned	_collActivityIndices[],
	const uint16_t	_collInteractPropIDs[],
	const unsigned	_collSrcIDs[],
	const unsigned	_collDstIDs[],
	const double	_collEquivMasses[],
	const double	_collEquivRadii[],
	const CVector3	_collContactVectors[],

	CVector3 _collTangOverlaps[],
	CVector3 _collTotalForces[]
)
{
	for (unsigned iActivColl = blockIdx.x * blockDim.x + threadIdx.x; iActivColl < *_collActiveCollisionsNum; iActivColl += blockDim.x * gridDim.x)
	{
		const unsigned       iColl                 = _collActivityIndices[iActivColl];
		const unsigned       iPart1                = _collSrcIDs[iColl];
		const unsigned       iPart2                = _collDstIDs[iColl];
		const SInteractProps prop                  = _interactProps[_collInteractPropIDs[iColl]];
		const double         radius1               = _partRadii[iPart1];
		const double         radius2               = _partRadii[iPart2];
		const double         equivRadius           = _collEquivRadii[iColl];
		const CVector3       contactVector         = _collContactVectors[iColl];

		const CVector3 normVector = contactVector.Normalized();

		const double surfaceDistance = (_partCoords[iPart2] - _partCoords[iPart1]).Length() - radius1 - radius2;

		constexpr double Dmin = 7.5e-10;
		const double hamakerConstant = 24 * PI * Dmin * Dmin * prop.dEquivSurfaceEnergy;
		double VdWForceLen;
		if (surfaceDistance <= Dmin)
			VdWForceLen = hamakerConstant * (2 * equivRadius) / (12 * Dmin*Dmin);
		else
			VdWForceLen = hamakerConstant * (2 * equivRadius) / (12 * pow(surfaceDistance + Dmin, 2.0));

		if (surfaceDistance < 0) // contact between surfaces
		{
			const double equivMass        = _collEquivMasses[iColl];
			const CVector3 angVel1        = _partAnglVels[iPart1];
			const CVector3 angVel2        = _partAnglVels[iPart2];
			const CVector3 tangOverlapOld = _collTangOverlaps[iColl];

			const CVector3 rc1 = contactVector * ( radius1 / (radius1 + radius2));
			const CVector3 rc2 = contactVector * (-radius2 / (radius1 + radius2));

			const double normOverlap = fabs(surfaceDistance);

			// normal and tangential relative velocity
			const CVector3 relVel = _partVels[iPart2] + angVel2 * rc2 - (_partVels[iPart1] + angVel1 * rc1);
			const double   normRelVelLen = DotProduct(normVector, relVel);
			const CVector3 normRelVel = normRelVelLen * normVector;
			const CVector3 tangRelVel = relVel - normRelVel;

			// radius of the contact area
			const double contactAreaRadius = sqrt(equivRadius * normOverlap);

			// normal force with damping
			const double Kn = 2 * prop.dEquivYoungModulus * contactAreaRadius;
			const double normContactForceLen = -normOverlap * Kn * 2. / 3.;
			const double normDampingForceLen = -_2_SQRT_5_6 * prop.dAlpha * normRelVelLen * sqrt(Kn * equivMass);
			const CVector3 normForce = normVector * (normContactForceLen + normDampingForceLen + VdWForceLen);

			// rotate old tangential overlap
			CVector3 tangOverlapRot = tangOverlapOld - normVector * DotProduct(normVector, tangOverlapOld);
			if (tangOverlapRot.IsSignificant())
				tangOverlapRot *= tangOverlapOld.Length() / tangOverlapRot.Length();
			// calculate new tangential overlap
			CVector3 tangOverlap = tangOverlapRot + tangRelVel * _timeStep;

			// tangential force with damping
			const double Kt = 8 * prop.dEquivShearModulus * contactAreaRadius;
			const CVector3 tangShearForce = tangOverlap * Kt;
			const CVector3 tangDampingForce = tangRelVel * (-_2_SQRT_5_6 * prop.dAlpha * sqrt(Kt * equivMass));

			// check slipping condition and calculate total tangential force
			CVector3 tangForce;
			const double tangShearForceLen = tangShearForce.Length();
			const double frictionForceLen = prop.dSlidingFriction * fabs(normContactForceLen + normDampingForceLen + VdWForceLen);
			if (tangShearForceLen > frictionForceLen)
			{
				tangForce = tangShearForce * frictionForceLen / tangShearForceLen;
				tangOverlap = tangForce / Kt;
			}
			else
				tangForce = tangShearForce + tangDampingForce;

			// rolling torque
			const CVector3 rollingTorque1 = angVel1.IsSignificant() ? angVel1 * (-prop.dRollingFriction * fabs(normContactForceLen) * radius1 / angVel1.Length()) : CVector3{ 0 };
			const CVector3 rollingTorque2 = angVel2.IsSignificant() ? angVel2 * (-prop.dRollingFriction * fabs(normContactForceLen) * radius2 / angVel2.Length()) : CVector3{ 0 };

			// final forces and moments
			const CVector3 totalForce = normForce + tangForce;
			const CVector3 moment1 = normVector * tangForce * radius1 + rollingTorque1;
			const CVector3 moment2 = normVector * tangForce * radius2 + rollingTorque2;

			// store results in collision
			_collTangOverlaps[iColl] = tangOverlap;
			_collTotalForces[iColl]  = totalForce;

			// apply moments and forces
			CUDA_VECTOR3_ATOMIC_ADD(_partForces[iPart1],  totalForce);
			CUDA_VECTOR3_ATOMIC_SUB(_partForces[iPart2] , totalForce);
			CUDA_VECTOR3_ATOMIC_ADD(_partMoments[iPart1], moment1);
			CUDA_VECTOR3_ATOMIC_ADD(_partMoments[iPart2], moment2);
		}
		else
		{
			// final forces and moments
			const CVector3 totalForce = normVector * VdWForceLen;

			// store results in collision
			_collTotalForces[iColl] = totalForce;

			// apply moments and forces
			CUDA_VECTOR3_ATOMIC_ADD(_partForces[iPart1], totalForce);
			CUDA_VECTOR3_ATOMIC_SUB(_partForces[iPart2], totalForce);
		}
	}
}